#include "hip/hip_runtime.h"
#include "Discretization.hpp"

Discretization::Discretization(int imax, int jmax, double dx, double dy,
                               double gamma) {
  double onedx = 1 / dx;
  double onedy = 1 / dy;
  hipMemcpyToSymbol(HIP_SYMBOL(_imax), &imax, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(_jmax), &jmax, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(_dx), &dx, sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(_dy), &dy, sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(_gamma), &gamma, sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(_one_dx), &onedx, sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(_one_dy), &onedy, sizeof(double));
}

__device__ double Discretization::convection_u(const double *U, const double *V,
                                               int i, int j) {

  int idx = _imax * j + i;
  int idx_right = _imax * j + (i + 1);
  int idx_left = _imax * j + (i - 1);

  double term1 =
      (1 / _dx) * (interpolate(U, i, j, 1, 0) * interpolate(U, i, j, 1, 0) -
                   interpolate(U, i, j, -1, 0) * interpolate(U, i, j, -1, 0)) +
      (_gamma / _dx) *
          (fabs(interpolate(U, i, j, 1, 0)) * (U[idx] - U[idx_right]) / 2 -
           fabs(interpolate(U, i, j, -1, 0)) * (U[idx_left] - U[idx]) / 2);

  int idx_top = _imax * (j + 1) + i;
  int idx_bottom = _imax * (j - 1) + i;
  double term2 =
      (1 / _dy) *
          (interpolate(V, i, j, 1, 0) * interpolate(U, i, j, 0, 1) -
           interpolate(V, i, j - 1, 1, 0) * interpolate(U, i, j, 0, -1)) +
      (_gamma / _dy) *
          (fabs(interpolate(V, i, j, 1, 0)) * (U[idx] - U[idx_top]) / 2 -
           fabs(interpolate(V, i, j - 1, 1, 0)) * (U[idx_bottom] - U[idx]) / 2);
  return term1 + term2;
}

__device__ double Discretization::convection_uSharedMem(const double *U,
                                                        const double *V, int i,
                                                        int j, int imax) {

  int idx = imax * j + i;
  int idx_right = idx + 1;//imax * j + (i + 1);
  int idx_left = idx - 1;//imax * j + (i - 1);

  double term1 = _one_dx * (interpolateSharedMem(U, i, j, 1, 0, imax) *
                                interpolateSharedMem(U, i, j, 1, 0, imax) -
                            interpolateSharedMem(U, i, j, -1, 0, imax) *
                                interpolateSharedMem(U, i, j, -1, 0, imax)) +
                 _gamma * _one_dx *
                     (fabs(interpolateSharedMem(U, i, j, 1, 0, imax)) *
                          (U[idx] - U[idx_right]) * 0.5 -
                      fabs(interpolateSharedMem(U, i, j, -1, 0, imax)) *
                          (U[idx_left] - U[idx]) * 0.5);

  int idx_top = idx + imax; //imax * (j + 1) + i;
  int idx_bottom = idx - imax;//imax * (j - 1) + i;
  double term2 = _one_dy * (interpolateSharedMem(V, i, j, 1, 0, imax) *
                                interpolateSharedMem(U, i, j, 0, 1, imax) -
                            interpolateSharedMem(V, i, j - 1, 1, 0, imax) *
                                interpolateSharedMem(U, i, j, 0, -1, imax)) +
                 _gamma * _one_dy *
                     (fabs(interpolateSharedMem(V, i, j, 1, 0, imax)) *
                          (U[idx] - U[idx_top]) * 0.5 -
                      fabs(interpolateSharedMem(V, i, j - 1, 1, 0, imax)) *
                          (U[idx_bottom] - U[idx]) * 0.5);
  return term1 + term2;
}

__device__ double Discretization::convection_v(const double *U, const double *V,
                                               int i, int j) {
  int idx = _imax * j + i;
  int idx_right = _imax * j + (i + 1);
  int idx_left = _imax * j + (i - 1);
  int idx_top = _imax * (j + 1) + i;
  int idx_bottom = _imax * (j - 1) + i;

  double term1 =
      (1 / _dy) * (interpolate(V, i, j, 0, 1) * interpolate(V, i, j, 0, 1) -
                   interpolate(V, i, j, 0, -1) * interpolate(V, i, j, 0, -1)) +
      (_gamma / _dy) *
          (fabs(interpolate(V, i, j, 0, 1)) * (V[idx] - V[idx_top]) / 2 -
           fabs(interpolate(V, i, j, 0, -1)) * (V[idx_bottom] - V[idx]) / 2);

  double term2 =
      (1 / _dx) *
          (interpolate(U, i, j, 0, 1) * interpolate(V, i, j, 1, 0) -
           interpolate(U, i - 1, j, 0, 1) * interpolate(V, i, j, -1, 0)) +
      (_gamma / _dx) *
          (fabs(interpolate(U, i, j, 0, 1)) * (V[idx] - V[idx_right]) / 2 -
           fabs(interpolate(U, i - 1, j, 0, 1)) * (V[idx_left] - V[idx]) / 2);

  return term1 + term2;
}

__device__ double Discretization::convection_vSharedMem(const double *U,
                                                        const double *V, int i,
                                                        int j, int imax) {
  int idx = imax * j + i;
  int idx_right = idx + 1;//imax * j + (i + 1);
  int idx_left = idx - 1;//imax * j + (i - 1);
  int idx_top = idx + imax; //imax * (j + 1) + i;
  int idx_bottom = idx - imax; //imax * (j - 1) + i;

  double term1 = _one_dy * (interpolateSharedMem(V, i, j, 0, 1, imax) *
                                interpolateSharedMem(V, i, j, 0, 1, imax) -
                            interpolateSharedMem(V, i, j, 0, -1, imax) *
                                interpolateSharedMem(V, i, j, 0, -1, imax)) +
                 _gamma * _one_dy *
                     (fabs(interpolateSharedMem(V, i, j, 0, 1, imax)) *
                          (V[idx] - V[idx_top]) * 0.5 -
                      fabs(interpolateSharedMem(V, i, j, 0, -1, imax)) *
                          (V[idx_bottom] - V[idx]) * 0.5);

  double term2 = _one_dx * (interpolateSharedMem(U, i, j, 0, 1, imax) *
                                interpolateSharedMem(V, i, j, 1, 0, imax) -
                            interpolateSharedMem(U, i - 1, j, 0, 1, imax) *
                                interpolateSharedMem(V, i, j, -1, 0, imax)) +
                 _gamma * _one_dx *
                     (fabs(interpolateSharedMem(U, i, j, 0, 1, imax)) *
                          (V[idx] - V[idx_right]) * 0.5 -
                      fabs(interpolateSharedMem(U, i - 1, j, 0, 1, imax)) *
                          (V[idx_left] - V[idx]) * 0.5);

  return term1 + term2;
}

__device__ double Discretization::convection_T(const double *U, const double *V,
                                               const double *T, int i, int j) {
  int idx = _imax * j + i;
  int idx_right = _imax * j + (i + 1);
  int idx_left = _imax * j + (i - 1);
  double term1 =
      (1 / (2 * _dx)) * (U[idx] * (T[idx] + T[idx_right]) -
                         U[idx_left] * (T[idx_left] + T[idx])) +
      (_gamma / (2 * _dx)) * (fabs(U[idx]) * (T[idx] - T[idx_right]) -
                              fabs(U[idx_left]) * (T[idx_left] - T[idx]));

  int idx_top = _imax * (j + 1) + i;
  int idx_bottom = _imax * (j - 1) + i;
  double term2 =
      (1 / (2 * _dy)) * (V[idx] * (T[idx] + T[idx_top]) -
                         V[idx_bottom] * (T[idx_bottom] + T[idx])) +
      (_gamma / (2 * _dy)) * (fabs(V[idx]) * (T[idx] - T[idx_top]) -
                              fabs(V[idx_bottom]) * (T[idx_bottom] - T[idx]));
  return term1 + term2;
}

__device__ double Discretization::convection_TSharedMem(const double *U,
                                                        const double *V,
                                                        const double *T, int i,
                                                        int j, int imax) {
  int idx = imax * j + i;
  int idx_right = idx + 1;//imax * j + (i + 1);
  int idx_left = idx - 1;//imax * j + (i - 1);
  double term1 = 0.5 * _one_dx *
                     (U[idx] * (T[idx] + T[idx_right]) -
                      U[idx_left] * (T[idx_left] + T[idx])) +
                 _gamma * 0.5 * _one_dx *
                     (fabs(U[idx]) * (T[idx] - T[idx_right]) -
                      fabs(U[idx_left]) * (T[idx_left] - T[idx]));

  int idx_top = idx + imax;//imax * (j + 1) + i;
  int idx_bottom = idx - imax;//imax * (j - 1) + i;
  double term2 = 0.5 * _one_dy *
                     (V[idx] * (T[idx] + T[idx_top]) -
                      V[idx_bottom] * (T[idx_bottom] + T[idx])) +
                 _gamma * 0.5 * _one_dy *
                     (fabs(V[idx]) * (T[idx] - T[idx_top]) -
                      fabs(V[idx_bottom]) * (T[idx_bottom] - T[idx]));
  return term1 + term2;
}

__device__ double Discretization::diffusion(const double *A, int i, int j) {
  int idx = _imax * j + i;
  int idx_right = _imax * j + i + 1;
  int idx_left = _imax * j + i - 1;
  double term1 = (A[idx_right] - 2 * A[idx] + A[idx_left]) / (_dx * _dx);

  int idx_top = _imax * (j + 1) + i;
  int idx_bottom = _imax * (j - 1) + i;

  double term2 = (A[idx_top] - 2 * A[idx] + A[idx_bottom]) / (_dy * _dy);
  return term1 + term2;
}

__device__ double Discretization::diffusionSharedMem(const double *A, int i,
                                                     int j, int imax) {
  int idx = imax * j + i;
  int idx_right = idx + 1;//imax * j + i + 1;
  int idx_left = idx - 1;//imax * j + i - 1;
  double term1 = (A[idx_right] - 2 * A[idx] + A[idx_left]) * _one_dx * _one_dx;

  int idx_top = idx + imax;//imax * (j + 1) + i;
  int idx_bottom = idx - imax;//imax * (j - 1) + i;

  double term2 = (A[idx_top] - 2 * A[idx] + A[idx_bottom]) * _one_dy * _one_dy;
  return term1 + term2;
}

__device__ double Discretization::laplacian(const double *P, int i, int j) {
  int idx = _imax * j + i;
  int idx_right = _imax * j + i + 1;
  int idx_left = _imax * j + i - 1;
  int idx_top = _imax * (j + 1) + i;
  int idx_bottom = _imax * (j - 1) + i;

  double result = (P[idx_right] - 2.0 * P[idx] + P[idx_left]) / (_dx * _dx) +
                  (P[idx_top] - 2.0 * P[idx] + P[idx_bottom]) / (_dy * _dy);

  return result;
}

__device__ double Discretization::laplacianSharedMem(const double *P, int i,
                                                     int j, int imax) {
  int idx = imax * j + i;
  int idx_right = idx+1;//imax * j + i + 1;
  int idx_left = idx-1;//imax * j + i - 1;
  int idx_top = idx+imax;//imax * (j + 1) + i;
  int idx_bottom = idx-imax;//imax * (j - 1) + i;

  double result =
      (P[idx_right] - 2.0 * P[idx] + P[idx_left]) * _one_dx * _one_dx +
      (P[idx_top] - 2.0 * P[idx] + P[idx_bottom]) * _one_dy * _one_dy;

  return result;
}

__device__ double Discretization::sor_helper(const double *P, int i, int j) {
  // int idx = _imax * j + i;
  int idx_right = _imax * j + i + 1;
  int idx_left = _imax * j + i - 1;
  int idx_top = _imax * (j + 1) + i;
  int idx_bottom = _imax * (j - 1) + i;

  double result = (P[idx_right] + P[idx_left]) / (_dx * _dx) +
                  (P[idx_top] + P[idx_bottom]) / (_dy * _dy);

  return result;
}

__device__ double Discretization::sor_helperSharedMem(const double *P, int i,
                                                      int j, int imax) {
  int idx = imax * j + i;
  int idx_right = idx + 1;//imax * j + i + 1;
  int idx_left = idx -1;//imax * j + i - 1;
  int idx_top = idx+imax;//imax * (j + 1) + i;
  int idx_bottom = idx-imax;//imax * (j - 1) + i;
  // double one_dy2 = _one_dy * _one_dy;
  // double one_dx2 = _one_dx * _one_dx;

  double result = (P[idx_right] + P[idx_left]) / (_dx * _dx) +
                  (P[idx_top] + P[idx_bottom]) / (_dy * _dy);

  return result;
}

__device__ double Discretization::interpolate(const double *A, int i, int j,
                                              int i_offset, int j_offset) {
  int idx = _imax * j + i;
  int idxOffset = _imax * (j + j_offset) + i + i_offset;

  return 0.5 * (A[idx] + A[idxOffset]);
}

__device__ double Discretization::interpolateSharedMem(const double *A, int i,
                                                       int j, int i_offset,
                                                       int j_offset, int imax) {
  int idx = imax * j + i;
  int idxOffset = idx + imax*j_offset + i_offset; //imax * (j + j_offset) + i + i_offset;

  return 0.5 * (A[idx] + A[idxOffset]);
}
