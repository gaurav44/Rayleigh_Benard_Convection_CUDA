#include "hip/hip_runtime.h"
#include "Simulation.hpp"
#include "thrust/device_vector.h"

__global__ void U_kernel_call(double *U, const double *F, const double *P,
                              double dx, int imax, double jmax, double dt) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  if (i > 0 && j > 0 && i < imax - 2 && j < jmax - 1) {
    int idx = imax * j + i;
    int idxRight = imax * j + i + 1;
    U[idx] = F[idx] - dt * (P[idxRight] - P[idx]) / dx;
  }
}

void U_kernel(Matrix &U, const Matrix &F, const Matrix &P,
              const Domain &domain) {
  dim3 threadsPerBlock(16, 16);
  dim3 numBlocks((domain.imax + 2 + threadsPerBlock.x - 1) / threadsPerBlock.x,
                 (domain.jmax + 2 + threadsPerBlock.y - 1) / threadsPerBlock.y);

  double *d_U = thrust::raw_pointer_cast(U.d_container.data());
  const double *d_P = thrust::raw_pointer_cast(P.d_container.data());
  const double *d_F = thrust::raw_pointer_cast(F.d_container.data());

  U_kernel_call<<<numBlocks, threadsPerBlock>>>(
      d_U, d_F, d_P, domain.dx, domain.imax + 2, domain.jmax + 2, domain.dt);
  hipDeviceSynchronize();
}

__global__ void V_kernel_call(double *V, const double *G, const double *P,
                              double dy, int imax, double jmax, double dt) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  if (i > 0 && j > 0 && i < imax - 1 && j < jmax - 2) {
    int idx = imax * j + i;
    int idxTop= imax * (j + 1) + i;
    V[idx] = G[idx] - dt * (P[idxTop] - P[idx]) / dy;
  }
}

void V_kernel(Matrix &V, const Matrix &G, const Matrix &P,
              const Domain &domain) {
  dim3 threadsPerBlock(16, 16);
  dim3 numBlocks((domain.imax + 2 + threadsPerBlock.x - 1) / threadsPerBlock.x,
                 (domain.jmax + 2 + threadsPerBlock.y - 1) / threadsPerBlock.y);

  double *d_V = thrust::raw_pointer_cast(V.d_container.data());
  const double *d_P = thrust::raw_pointer_cast(P.d_container.data());
  const double *d_G = thrust::raw_pointer_cast(G.d_container.data());

  V_kernel_call<<<numBlocks, threadsPerBlock>>>(
      d_V, d_G, d_P, domain.dy, domain.imax + 2, domain.jmax + 2, domain.dt);
  hipDeviceSynchronize();
}
