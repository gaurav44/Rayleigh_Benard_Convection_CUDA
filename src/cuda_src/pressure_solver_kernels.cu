#include "hip/hip_runtime.h"
#include "block_sizes.hpp"
#include "cuda_utils.hpp"
#include "pressure_solver_kernels.hpp"
#include "discretization.hpp"
#include <vector>

//#define BLOCK_SIZE 16
namespace PressureSolverKernels {
__global__ void SOR_kernel_call(double *P, const double *RS, int imax,
                                double jmax, double omg, double coeff,
                                int color) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  //  double coeff = omg / (2.0 * (1.0 / (dx * dx) + 1.0 / (dy * dy)));

  if (i > 0 && j > 0 && i < imax - 1 && j < jmax - 1 && (i + j) % 2 == color) {
    int idx = imax * j + i;
    P[idx] = (1.0 - omg) * P[idx] +
             coeff * (Discretization::sor_helper(P, i, j) - RS[idx]);
  }
}

__global__ void SORKernelShared(double *P, const double *RS, int imax,
                                      double jmax, double omg, double coeff,
                                      int color) {
  int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
  int j = blockIdx.y * blockDim.y + threadIdx.y + 1;

  int global_idx = j * imax + i;
  __shared__ double shared_P[(BLOCK_SIZE_SOR + 2) * (BLOCK_SIZE_SOR + 2)];

  int local_i = threadIdx.x + 1;
  int local_j = threadIdx.y + 1;
  int local_idx = local_j * (blockDim.x + 2) + local_i;

  // load the central part into shared memory
  if (local_i > 0 && local_j > 0 && local_i < blockDim.x + 1 &&
      local_j < blockDim.y + 1)
    shared_P[local_idx] = P[global_idx];

  // Left Halo
  if (threadIdx.x == 0 && i > 0)
    shared_P[local_idx - 1] = P[global_idx - 1];

  // Right Halo
  if ((threadIdx.x == blockDim.x - 1 || blockIdx.x == gridDim.x - 1) &&
      i < imax - 1)
    shared_P[local_idx + 1] = P[global_idx + 1];

  // Bottom Halo
  if (threadIdx.y == 0 && j > 0)
    shared_P[local_idx - blockDim.x - 2] = P[global_idx - imax];

  // Top Halo
  if ((threadIdx.y == blockDim.y - 1 || blockDim.y == gridDim.y - 1) &&
      j < jmax - 1)
    shared_P[local_idx + blockDim.x + 2] = P[global_idx + imax];

  __syncthreads();

  if (i < imax - 1 && j < jmax - 1 && (i + j) % 2 == color) {
    shared_P[local_idx] =
        (1.0 - omg) * shared_P[local_idx] +
        coeff * (Discretization::sor_helperSharedMem(shared_P, local_i, local_j,
                                                     blockDim.x + 2) -
                 RS[global_idx]);
    P[global_idx] = shared_P[local_idx];
  }
}

//__global__ void Residual_kernel_call(const double *P, const double *RS,
//                                     int imax, double jmax, double *residual)
//                                     {
//  int i = blockIdx.x * blockDim.x + threadIdx.x;
//  int j = blockIdx.y * blockDim.y + threadIdx.y;
//
//  if (i > 0 && j > 0 && i < imax - 1 && j < jmax - 1) {
//    int idx = imax * j + i;
//    double val = Discretization::laplacian(P, i, j) - RS[idx];
//    atomicAdd(residual, (val * val));
//  }
//}

__global__ void residualKernelShared(const double *P, const double *RS,
                                           int imax, double jmax,
                                           double *residual_results) {
  int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
  int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
  int global_idx = j * imax + i;

  __shared__ double shared_P[(BLOCK_SIZE_RES + 2) * (BLOCK_SIZE_RES + 2)];
  __shared__ double shared_val[BLOCK_SIZE_RES * BLOCK_SIZE_RES];
  int shared_index = threadIdx.x * blockDim.y + threadIdx.y;
  shared_val[shared_index] = 0.0;

  int local_i = threadIdx.x + 1;
  int local_j = threadIdx.y + 1;
  int local_idx = local_j * (blockDim.x + 2) + local_i;

  // load the central part into shared memory
  if (local_i > 0 && local_j > 0 && local_i < blockDim.x + 1 &&
      local_j < blockDim.y + 1) {
    shared_P[local_idx] = P[global_idx];
  }

  // Left Halo
  if (threadIdx.x == 0 && i > 0) {
    shared_P[local_idx - 1] = P[global_idx - 1];
  }

  // Right Halo
  if ((threadIdx.x == blockDim.x - 1 || blockIdx.x == gridDim.x - 1) &&
      i < imax - 1) {
    shared_P[local_idx + 1] = P[global_idx + 1];
  }

  // Bottom Halo
  if (threadIdx.y == 0 && j > 0) {
    shared_P[local_idx - blockDim.x - 2] = P[global_idx - imax];
  }

  // Top Halo
  if ((threadIdx.y == blockDim.y - 1 || blockIdx.y == gridDim.y - 1) &&
      j < jmax - 1) {
    shared_P[local_idx + blockDim.x + 2] = P[global_idx + imax];
  }

  __syncthreads();

  if (i < imax - 1 && j < jmax - 1) {
    double val = Discretization::laplacianSharedMem(shared_P, local_i, local_j,
                                                    blockDim.x + 2) -
                 RS[global_idx];
    shared_val[shared_index] = val * val;
    __syncthreads();

    // Perform reduction in shared memory
    for (int s = (blockDim.x * blockDim.y) / 2; s > 0; s /= 2) {
      if (shared_index < s) {
        shared_val[shared_index] =
            shared_val[shared_index] + shared_val[shared_index + s];
      }
      __syncthreads(); // Synchronize after each reduction step
    }
    // atomicAdd(residual_results, (val * val));
  }

  // Write the result of this block's max to global memory
  if (threadIdx.x == 0 && threadIdx.y == 0) {
    residual_results[blockIdx.x * gridDim.y + blockIdx.y] = shared_val[0];
  }
}

double calculatePressureKernel(Matrix &P, const Matrix &RS, const Domain &domain,
                             double omg, double *d_rlocBlock, std::vector<double>& h_rlocBlock) {
  dim3 threadsPerBlock(BLOCK_SIZE_SOR, BLOCK_SIZE_SOR);
  dim3 numBlocks((domain.imax + 2 + threadsPerBlock.x - 1) / threadsPerBlock.x,
                 (domain.jmax + 2 + threadsPerBlock.y - 1) / threadsPerBlock.y);

  size_t shared_mem_sor =
      (threadsPerBlock.x + 2) * (threadsPerBlock.y + 2) * 1 * sizeof(double);

  double coeff =
      omg /
      (2.0 * (1.0 / (domain.dx * domain.dx) + 1.0 / (domain.dy * domain.dy)));

  SORKernelShared<<<numBlocks, threadsPerBlock, shared_mem_sor>>>(
      thrust::raw_pointer_cast(P.d_container.data()),
      thrust::raw_pointer_cast(RS.d_container.data()), domain.imax + 2,
      domain.jmax + 2, omg, coeff, 0);
  CHECK(hipGetLastError());

  SORKernelShared<<<numBlocks, threadsPerBlock, shared_mem_sor>>>(
      thrust::raw_pointer_cast(P.d_container.data()),
      thrust::raw_pointer_cast(RS.d_container.data()), domain.imax + 2,
      domain.jmax + 2, omg, coeff, 1);
  CHECK(hipGetLastError());
 
  double res = 0.0;
  //double h_rlocBlock[numBlocks.x * numBlocks.y];
  //std::vector<double> h_rlocBlock(numBlocks.x*numBlocks.y);
  double h_rloc = 0.0;

  size_t shared_mem_residual =
      (threadsPerBlock.x + 2) * (threadsPerBlock.y + 2) * 1 * sizeof(double) +
      threadsPerBlock.x * threadsPerBlock.y;

  residualKernelShared<<<numBlocks, threadsPerBlock,
                               shared_mem_residual>>>(
      thrust::raw_pointer_cast(P.d_container.data()),
      thrust::raw_pointer_cast(RS.d_container.data()), domain.imax + 2,
      domain.jmax + 2, d_rlocBlock);

  CHECK(hipGetLastError());

  CHECK(hipMemcpy(h_rlocBlock.data(), d_rlocBlock,
                   numBlocks.x * numBlocks.y * sizeof(double),
                   hipMemcpyDeviceToHost));

  // Find the maximum in the result array
  for (int i = 0; i < numBlocks.x * numBlocks.y; ++i) {
    h_rloc = h_rloc + h_rlocBlock[i];
  }

  res = h_rloc / (domain.imax * domain.jmax);
  res = std::sqrt(res);

  return res;
}
} // namespace PressureSolverKernels
